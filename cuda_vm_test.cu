#include <hip/hip_runtime.h>

#include <sys/mman.h>
#include <sys/stat.h>
#include <sys/shm.h>
#include <fcntl.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include <iostream>

static inline void
checkDrvError(hipError_t res, const char *tok, const char *file, unsigned line)
{
    if (res != hipSuccess) {
        const char *errStr = NULL;
        (void)hipDrvGetErrorString(res, &errStr);
        std::cerr << file << ':' << line << ' ' << tok
                  << "failed (" << (unsigned)res << "): " << errStr << std::endl;
        abort();
    }
}

#define CHECK_DRV(x) checkDrvError(x, #x, __FILE__, __LINE__);

__global__ void square(int * array) {
    int tid = threadIdx.x;
    array[tid] = array[tid] * array[tid];
}

class Thing {
    public:
    Thing(int i) {
        x = i;
        y = i * i;
        z = i * i * i;
    }

private:
    int x,y,z;
};

int main(int argc, char **argv) {
    void * ptr = malloc(12);

    Thing * t = new(ptr) Thing(4);



    int id = shmget(854, 0x400000, IPC_CREAT | 0600);
    if (id == -1) {
        std::cout << "shmget: " << errno << std::endl;
        return -1;
    }
    void * addr = shmat(id, NULL, 0);
    if (addr == (void*)-1) {
        std::cout << "shmat: " << errno << std::endl;
        return -1;
    }
    if (shmctl(id, IPC_RMID, NULL) == -1)
        std::cout << "shmctl: " << errno << std::endl;
    

    std::cout << "Pagesize: " << getpagesize() << std::endl;

    std::cout << "Catch me at " << getpid() << std::endl;

    std::cout << "Doing cuda things" << std::endl;
    CHECK_DRV(hipInit(0));

    hipMemAllocationProp props = {};
    props.type = hipMemAllocationTypePinned;
    props.location.type = hipMemLocationTypeDevice;
    props.location.id = 0;

    hipDeviceptr_t d_addr;
    hipDeviceptr_t d_hint = 0x00004dead000000;
    CHECK_DRV(hipMemAddressReserve(&d_addr, 0x2000000, 0x100, 0, 0ULL));

    hipMemGenericAllocationHandle_t handle;
    CHECK_DRV(hipMemCreate(&handle, 0x200000, &props, 0));

    CHECK_DRV(hipMemMap(d_addr + 0x400000, 0x200000, 0, handle, 0));

    hipMemAccessDesc accessDesc;
    accessDesc.location = props.location;
    accessDesc.flags = hipMemAccessFlagsProtReadWrite;
    CHECK_DRV(hipMemSetAccess(d_addr + 0x400000, 0x200000, &accessDesc, 1));

    int buffer[] = {1,2,3,4,5,6,7,8};
    hipMemcpy((void*)d_addr, buffer, 8 * sizeof(int), hipMemcpyHostToDevice);

    square<<<1, 8>>>((int*)d_addr);
    hipDeviceSynchronize();

    hipMemcpy(buffer, (void*)d_addr, 8 * sizeof(int), hipMemcpyDeviceToHost);
    for(int i = 0; i < 8; i++) {
        std::cout << buffer[i] << " ";
    }


    std::cout << "Shared memory stuff" << std::endl;
    void * adj_addr = shmat(id, (void*)(d_addr), SHM_RDONLY | SHM_REMAP);
    
    *(int*)addr = 69420;
    if (*(int*)adj_addr == 69420) {
        std::cout << "Mapping successful" << std::endl;
    }

    CHECK_DRV(hipMemUnmap(d_addr + 0x400000, 0x200000));
    CHECK_DRV(hipMemRelease(handle));
    CHECK_DRV(hipMemAddressFree(d_addr, 0x2000000));

    shmdt(addr);
    shmdt(adj_addr);
    close(id);



    return 0;
}