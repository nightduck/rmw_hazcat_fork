#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "rmw_hazcat_cpp/allocators/cpu_pool_allocator.hpp"
#include <iostream>

template<typename T>
class BaseClass {
public:
    int thing;
    double otherThing;

    template<class AllocT>
    static AllocT * static_fn(int i);

    virtual void virtual_fn(int i );
};

// static void static_fn(int i) {
//     std::cout << "int static_fn: " << i << std::endl;
// }

class DerivedClass : BaseClass<int> {
public:
    DerivedClass(int i) {
        thing = i;
        otherThing = i;
    }

    static void static_fn(int i) {
        std::cout << "int static_fn: " << i << std::endl;
    }

    void virtual_fn(int i ) override {
        thing = i;
    }
};

class SelfDestruct {
public:
    SelfDestruct() {
        std::cout << "Constructing at " << this << std::endl;
        i = 4;
        j = 6;
        free(this);
        return;
    }

    int i, j;
};

using AllocT = StaticPoolAllocator<long, 30>;

int main(int argc, char ** argv) {
    AllocT * cpu_alloc;
    void * ptr;

    // DerivedClass c = DerivedClass(4);
    // DerivedClass::static_fn(3);


    cpu_alloc = AllocT::create_shared_alloc();

    cpu_alloc->allocate();
    ptr = malloc(sizeof(long));
    void * copy = cpu_alloc->convert(ptr, sizeof(long), cpu_alloc);
    //static_deallocate<AllocT>(cpu_alloc, copy);
    AllocT::static_deallocate(cpu_alloc, copy);

    UnknownAllocator * uk = (UnknownAllocator*)cpu_alloc;
    uk = UnknownAllocator::map_shared_alloc(cpu_alloc->get_id());

    cpu_alloc->~StaticPoolAllocator();

    //cpu_alloc = HMAAllocator<CPU_Mem>::create_shared_alloc<AllocT>();

    //cpu_alloc = AllocT::create_shared_alloc<AllocT>(64);

    //FakeClass * fake = HMAAllocator<CPU_Mem>::create_shared_alloc<FakeClass>();


    return 0;
}