#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "rmw_hazcat_cpp/allocators/cpu_pool_allocator.hpp"


class FakeClass {
public:
    int thing;
    double otherThing;

    FakeClass() {
        thing = 5;
        otherThing = 5.6;
    }
};

using AllocT = StaticPoolAllocator<FakeClass, 30>;

int main(int argc, char ** argv) {
    AllocT * cpu_alloc;
    void * ptr;

    cpu_alloc = HMAAllocator<CPU_Mem>::create_shared_alloc<AllocT>();

    //cpu_alloc = AllocT::create_shared_alloc<AllocT>(64);

    //FakeClass * fake = HMAAllocator<CPU_Mem>::create_shared_alloc<FakeClass>();


    return 0;
}