#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include "rmw_hazcat_cpp/allocators/cpu_pool_allocator.hpp"
#include <iostream>
#include <cassert>

template<typename T>
class BaseClass {
public:
    int thing;
    double otherThing;

    template<class AllocT>
    static AllocT * static_fn(int i);

    virtual void virtual_fn(int i );
};

// static void static_fn(int i) {
//     std::cout << "int static_fn: " << i << std::endl;
// }

class DerivedClass : BaseClass<int> {
public:
    DerivedClass(int i) {
        thing = i;
        otherThing = i;
    }

    static void static_fn(int i) {
        std::cout << "int static_fn: " << i << std::endl;
    }

    void virtual_fn(int i ) override {
        thing = i;
    }
};

class SelfDestruct {
public:
    SelfDestruct() {
        std::cout << "Constructing at " << this << std::endl;
        i = 4;
        j = 6;
        free(this);
        return;
    }

    int i, j;
};

using AllocT = StaticPoolAllocator<long, 30>;

int main(int argc, char ** argv) {
    AllocT * cpu_alloc = AllocT::create_shared_alloc();

    assert(sizeof(int) == 4UL);
    assert(sizeof(long) == 8UL);
    assert(sizeof(void(*)(void*)) == 8UL);

    assert(cpu_alloc != nullptr);
    //TODO: Something with checking the shmem_id. EXPECT_EQ(((int*)cpu_alloc))

    uint8_t* ptr = (uint8_t*)cpu_alloc;
    int id = cpu_alloc->get_id();
    // assert(*(uint64_t*)(ptr+4) == (uint64_t)&AllocT::static_deallocate);
    // assert(*(uint64_t*)(ptr+12) == (uint64_t)&AllocT::static_remap);

    std::cout << "Pre destruction, ID is : " << cpu_alloc->get_id() << std::endl;
    cpu_alloc->~StaticPoolAllocator();
    std::cout << "Destructed" << std::endl;
    std::cout << "Post destruction, ID is : " << cpu_alloc->get_id() << std::endl;


    return 0;
}